#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/large_margin_inner_product_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void copy_label_score(const int M, const int N, const Dtype *label_data,
	const Dtype *top_data, Dtype *wx_data) {
	CUDA_KERNEL_LOOP(index, M) {
		wx_data[index] = top_data[index * N + static_cast<int>(label_data[index])];
	}
}

template <typename Dtype>
__global__ void cal_cos_mt(const int count, const unsigned int margin, const int *C_M_N, const Dtype *cos_t_data,
	Dtype *cos_mt_data) {
	CUDA_KERNEL_LOOP(index, count) {
		Dtype cos_t = cos_t_data[index];
		Dtype sin_t_2 = 1 - cos_t * cos_t;
		Dtype cos_mt = 0.;
		int flag = -1;
		for(int n = 0; n <= (margin / 2); ++n) {
			flag *= -1;
			cos_mt += flag * C_M_N[2 * n] * powf(cos_t, (margin - 2 * n)) * powf(sin_t_2, n);
		}
		cos_mt_data[index] = cos_mt;
	}
}

template <typename Dtype>
__global__ void LMForward(
  const int M, const int N, const float lambda,
  const Dtype *label_data, const Dtype *cos_mt_data, const int *k_data,
  const Dtype *abs_w_data, const Dtype *abs_x_data, Dtype *top_data) {
 
  CUDA_KERNEL_LOOP(index, M) {
    Dtype cos_mt = cos_mt_data[index];
    int k = k_data[index];
    int label = static_cast<int>(label_data[index]);
    Dtype abs_w = abs_w_data[index];
    Dtype abs_x = abs_x_data[index];
    top_data[N * index + label] =  (lambda * top_data[N * index + label] + 
    	abs_w * abs_x * ( powf(-1, k) * cos_mt - 2 * k )) / (1 + lambda);
  }
}

// use for compute gradient for weight 
template <typename Dtype>
__global__ void LMBackward_weight(
  const int M, const int N, const float lambda, const unsigned int margin,
  const Dtype *label_data, const int *C_M_N, const int *k_data, const Dtype *weight_data, 
  const Dtype *bottom_data, const Dtype *wx_data, const Dtype *abs_w_data, 
  const Dtype *abs_x_data, Dtype *top_data) {
 
  CUDA_KERNEL_LOOP(index, M) {
    int k = k_data[index];
    int label = static_cast<int>(label_data[index]);
    Dtype abs_w = abs_w_data[index];
    Dtype abs_x = abs_x_data[index];
    // |x| * |w|
    Dtype res_wx = abs_x * abs_w;
    Dtype x_data = bottom_data[index];
    // d(cos_mt) / d w
    Dtype dcos_mt_data = 0.;
    // compute when n == 0
    dcos_mt_data = C_M_N[0] * (margin * powf(wx_data[N * index + label], margin-1) * 
    				x_data / (powf(res_wx, margin -1)) - (margin-1) * powf(wx_data[N * index + label], margin) * 
    				weight_data[N * index + label] / powf(abs_w, margin + 1) / powf(abs_x, margin - 1));
    
    for (int n = 1; n <= margin / 2; ++n){
    	dcos_mt_data += C_M_N[2 * n] * ((margin - 2 * n -1) * powf(wx_data[N * index + label], margin-2*n) * 
    		weight_data[N * index + label] / powf(abs_w, margin - 2*n + 1) / powf(abs_x, margin - 2*n - 1) - 
    		(margin - 2 * n) * powf(wx_data[N * index + label], margin-2*n -1) * 
    		x_data / powf(res_wx, margin - 2*n - 1) + margin * powf(wx_data[N * index + label], margin - 2 * n + 1) * 
    		x_data / powf(res_wx, margin - 2*n + 1) - (margin - 2*n + 1)* powf(wx_data[N * index + label], margin) * 
    		weight_data[N * index + label] / powf(abs_w, margin + 1) / powf(abs_x, margin - 2*n + 1));
    }
    top_data[N * index + label] =  (lambda * top_data[N * index + label] + 
    	(powf(-1, k) * dcos_mt_data - 2 * k * abs_x * weight_data[N * index + label] / abs_w)) / (1 + lambda);
  }
}

// use for compute gradient for bottom data 
template <typename Dtype>
__global__ void LMBackward_data(
  const int M, const int N, const float lambda, const unsigned int margin,
  const Dtype *label_data, const int *C_M_N, const int *k_data, const Dtype *weight_data, 
  const Dtype *bottom_data, const Dtype *wx_data, const Dtype *abs_w_data, 
  const Dtype *abs_x_data, Dtype *top_data) {
 
  CUDA_KERNEL_LOOP(index, M) {
    int k = k_data[index];
    int label = static_cast<int>(label_data[index]);
    Dtype abs_w = abs_w_data[index];
    Dtype abs_x = abs_x_data[index];
    // |x| * |w|
    Dtype res_wx = abs_x * abs_w;
    Dtype x_data = bottom_data[index];
    // d(cos_mt) / d x
    Dtype dcos_mt_x = 0.;
    // compute when n == 0
    dcos_mt_x = C_M_N[0] * (margin * powf(wx_data[N * index + label], margin-1) * 
    				weight_data[N * index + label] / (powf(res_wx, margin -1)) - (margin-1) * powf(wx_data[N * index + label], margin) * 
    				x_data / powf(abs_w, margin - 1) / powf(abs_x, margin + 1));
    
    for (int n = 1; n <= margin / 2; ++n){
    	dcos_mt_x += C_M_N[2 * n] * ((margin - 2 * n -1) * powf(wx_data[N * index + label], margin-2*n) * 
    		x_data / powf(abs_w, margin - 2*n + 1) / powf(abs_x, margin - 2*n - 1) - 
    		(margin - 2 * n) * powf(wx_data[N * index + label], margin-2*n -1) * 
    		weight_data[N * index + label] / powf(res_wx, margin - 2*n - 1) + margin * powf(wx_data[N * index + label], margin - 2 * n + 1) * 
    		weight_data[N * index + label] / powf(res_wx, margin - 2*n + 1) - (margin - 2*n + 1)* powf(wx_data[N * index + label], margin) * 
    		x_data / powf(abs_w, margin - 2*n + 1) / powf(abs_x, margin + 1));
    }
    top_data[N * index + label] =  (lambda * top_data[N * index + label] + 
    	(powf(-1, k) * dcos_mt_x - 2 * k * abs_x * weight_data[N * index + label] / abs_w)) / (1 + lambda);
  }
}



template <typename Dtype>
void LargeMarginInnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* label_data = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
 
  // 普通fc层的计算
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         weight, bottom_data, (Dtype)0., top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans,
                          transpose_ ? CblasNoTrans : CblasTrans,
                          M_, N_, K_, (Dtype)1.,
                          bottom_data, weight, (Dtype)0., top_data);
  }
 
  const Dtype* label_cpu_data = bottom[1]->cpu_data();
 
  // w * x
  // 直接从前馈的结果中复制
  Dtype *wx_data = this->wx_.mutable_gpu_data();
  copy_label_score<Dtype><<<CAFFE_GET_BLOCKS(M_), CAFFE_CUDA_NUM_THREADS>>>(M_, N_, label_data, top_data, wx_data);
 
  // w * w
  Dtype *abs_w_data = this->abs_w_.mutable_cpu_data();
  for (int m = 0; m < M_; ++ m) {
    abs_w_data[m] = caffe_cpu_dot<Dtype>(
      K_,
      this->blobs_[0]->cpu_data() + static_cast<int>(label_cpu_data[m]) * K_,
      this->blobs_[0]->cpu_data() + static_cast<int>(label_cpu_data[m]) * K_
      );
  }
   // x * x
  Dtype *abs_x_data = this->abs_x_.mutable_cpu_data();
  for (int m = 0; m < M_; ++ m) {
    abs_x_data[m] = caffe_cpu_dot<Dtype>(
      K_, 
      bottom[0]->cpu_data() + m * K_,
      bottom[0]->cpu_data() + m * K_
      );
  }
 
  // abs_w, abs_x
  caffe_gpu_powx<Dtype>(M_, this->abs_w_.mutable_gpu_data(), 0.5, this->abs_w_.mutable_gpu_data());
  caffe_gpu_powx<Dtype>(M_, this->abs_x_.mutable_gpu_data(), 0.5, this->abs_x_.mutable_gpu_data());
 
  // cos_t = wx / (|x| * |w|)
  Dtype *cos_t_data = this->cos_t_.mutable_gpu_data();
  caffe_gpu_div<Dtype>(M_, wx_data, this->abs_x_.gpu_data(), cos_t_data);
  caffe_gpu_div<Dtype>(M_, cos_t_data, this->abs_w_.gpu_data(), cos_t_data);
 
  // cos(mt)
  cal_cos_mt<Dtype><<<CAFFE_GET_BLOCKS(M_), CAFFE_CUDA_NUM_THREADS>>>(
    M_, this->margin, 
    this->C_M_N_.gpu_data(), 
    this->cos_t_.gpu_data(),
    this->cos_mt_.mutable_gpu_data()
    );
  
  // k
  int *k_cpu_data = this->k_.mutable_cpu_data();
  const Dtype *cos_t_cpu_data = this->cos_t_.cpu_data();
  for (int m = 0; m < M_; ++ m) {
    for (int _k = 0; _k < this->cos_theta_bound_.count(); ++ _k) {
      if (this->cos_theta_bound_.cpu_data()[_k] < cos_t_cpu_data[m]) {
        k_cpu_data[m] = _k - 1;
        break;
      }
    }
  }
 
  // y
  LMForward<Dtype><<<CAFFE_GET_BLOCKS(M_), CAFFE_CUDA_NUM_THREADS>>>(
    M_, N_, this->lambda,
    label_data, this->cos_mt_.gpu_data(), this->k_.gpu_data(),
    this->abs_w_.gpu_data(), this->abs_x_.gpu_data(), top[0]->mutable_gpu_data());
}

template <typename Dtype>
void LargeMarginInnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {

	// w * x
	const Dtype* wx_data = this->wx_.gpu_data();
	// label
	const Dtype* label_data = bottom[1]->gpu_data();

	if (this->param_propagate_down_[0]) {
	    const Dtype* top_diff = top[0]->gpu_diff();
	    const Dtype* bottom_data = bottom[0]->gpu_data();
	    // Gradient with respect to weight
	    if (transpose_) {
	      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
	          K_, N_, M_,
	          (Dtype)1., bottom_data, top_diff,
	          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
	    } else {
	      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
	          N_, K_, M_,
	          (Dtype)1., top_diff, bottom_data,
	          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
	    }
	    LMBackward_weight<Dtype><<<CAFFE_GET_BLOCKS(M_), CAFFE_CUDA_NUM_THREADS>>>(
	    	M_, N_, this->lambda, this->margin,
	    	label_data, this->C_M_N_.gpu_data(), this->k_.gpu_data(),
	    	this->blobs_[0]->gpu_data(), bottom_data, wx_data,
	    	this->abs_w_.gpu_data(), this->abs_x_.gpu_data(), this->blobs_[0]->mutable_gpu_diff());
  }

    if (propagate_down[0]) {
	    const Dtype* top_diff = top[0]->gpu_diff();
	    const Dtype* bottom_data = bottom[0]->gpu_data();
	    // Gradient with respect to bottom data
	    if (transpose_) {
	      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
	          M_, K_, N_,
	          (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
	          (Dtype)0., bottom[0]->mutable_gpu_diff());
	    } else {
	      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
	          M_, K_, N_,
	         (Dtype)1., top_diff, this->blobs_[0]->gpu_data(),
	         (Dtype)0., bottom[0]->mutable_gpu_diff());
	    }

	    LMBackward_data<Dtype><<<CAFFE_GET_BLOCKS(M_), CAFFE_CUDA_NUM_THREADS>>>(
	    	M_, N_, this->lambda, this->margin,
	    	label_data, this->C_M_N_.gpu_data(), this->k_.gpu_data(),
	    	this->blobs_[0]->gpu_data(), bottom_data, wx_data,
	    	this->abs_w_.gpu_data(), this->abs_x_.gpu_data(), bottom[0]->mutable_gpu_diff());
	  }

}

INSTANTIATE_LAYER_GPU_FUNCS(LargeMarginInnerProductLayer);

} // namespace caffe